#include <cstdint>
#include <iostream>
#include <algorithm>

#include <hip/hip_runtime.h>

using namespace std;
__global__ void countValidCombinations(int* ans) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i <= 22 && j <= 22) {
        if (24 - i - j > 0) {
            if (i != j && j != 24 - i - j && i != 24 - i - j) {
                atomicAdd(ans, 1);
            }
        }
    }
}
int main() {
    int ans = 0;
    int* d_ans;

    hipError_t cudaStatus;

    // 分配设备内存
    cudaStatus = hipMalloc((void**)&d_ans, sizeof(int));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        return 1;
    }

    // 将主机数据拷贝到设备
    cudaStatus = hipMemcpy(d_ans, &ans, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        hipFree(d_ans);
        return 1;
    }

    // 定义CUDA线程块和网格大小
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((22 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (22 + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // 调用CUDA核函数
    countValidCombinations<<<numBlocks, threadsPerBlock>>>(d_ans);

    // 将设备数据拷贝回主机
    cudaStatus = hipMemcpy(&ans, d_ans, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        hipFree(d_ans);
        return 1;
    }

    cout << ans << endl;

    // 释放设备内存
    hipFree(d_ans);

    return 0;
}